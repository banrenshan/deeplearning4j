#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2019 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 06.02.2019
// @author raver119@gmail.com
//

#include <PointersManager.h>
#include <StringUtils.h>
//#include <graph/exceptions/cuda_exception.h>
#include <stdexcept>
#include <logger.h>
#include <memory/Workspace.h>

namespace nd4j {

//////////////////////////////////////////////////////////////////////////
PointersManager::PointersManager(nd4j::graph::LaunchContext *context, const std::string& funcName)  {
        _context  = context;
        _funcName = funcName;
}

//////////////////////////////////////////////////////////////////////////
void* PointersManager::replicatePointer(const void* src, const size_t numberOfBytes) {

	void* dst = nullptr;
	if (_context->getWorkspace() == nullptr) {
        hipError_t cudaResult = hipMalloc(reinterpret_cast<void **>(&dst), numberOfBytes);
        if (cudaResult != 0)
            //throw cuda_exception::build(_funcName + ": cannot allocate global memory on device!", cudaResult);
            throw std::runtime_error(_funcName + ": cannot allocate global memory on device!");
    } else {
	    dst = _context->getWorkspace()->allocateBytes(nd4j::memory::MemoryType::DEVICE, numberOfBytes);
	}

    if (_context != nullptr)
        hipMemcpyAsync(dst, src, numberOfBytes, hipMemcpyHostToDevice, *_context->getCudaStream());
    else
        hipMemcpy(dst, src, numberOfBytes, hipMemcpyHostToDevice);

    _pOnGlobMem.emplace_back(dst);
    
    return dst;
}

//////////////////////////////////////////////////////////////////////////
void PointersManager::synchronize() const {
    if (_context != nullptr) {
        hipError_t cudaResult = hipStreamSynchronize(*_context->getCudaStream());
        if (cudaResult != 0) {
            //throw cuda_exception::build(_funcName + ": cuda stream synchronization failed !", cudaResult);
            auto cd = nd4j::StringUtils::valueToString<int>((int) cudaResult);
            throw std::runtime_error(_funcName + ": cuda stream synchronization failed! Error code: " + cd);
        }
    } else {
        nd4j_printf("<%s> syncStream isn't possible: no stream set!", _funcName.c_str());
    }
}

//////////////////////////////////////////////////////////////////////////
PointersManager::~PointersManager() {
    
    for (auto& p :_pOnGlobMem)
        hipFree(p);
}

template <typename T>
static __global__ void _printDevContentOnDev(void* pDev, Nd4jLong len, int tid) {

    if(blockIdx.x * blockDim.x + threadIdx.x != tid)
        return;

    printf("device print out: \n");
    for(Nd4jLong i = 0; i < len; ++i)
        printf("%f, ", (double)reinterpret_cast<T*>(pDev)[i]);

    printf("\n");
}

////////////////////////////////////////////////////////////////////////
template<typename T>
void PointersManager::printDevContentOnDev(void* pDev, Nd4jLong len, int tid) {
    _printDevContentOnDev<T><<<512, 512, 1024, *graph::LaunchContext::defaultContext()->getCudaStream()>>>(pDev, len, tid);
    auto res = hipStreamSynchronize(*graph::LaunchContext::defaultContext()->getCudaStream());
    if (res != 0)
        throw std::runtime_error("PointersManager::printDevContentOnDev: hipStreamSynchronize failed!");
}
template void PointersManager::printDevContentOnDev<Nd4jLong>(void* pDev, Nd4jLong len, int tid);
template void PointersManager::printDevContentOnDev<int>(void* pDev, Nd4jLong len, int tid);
template void PointersManager::printDevContentOnDev<float>(void* pDev, Nd4jLong len, int tid);
template void PointersManager::printDevContentOnDev<double>(void* pDev, Nd4jLong len, int tid);

//BUILD_SINGLE_TEMPLATE(template void PointersManager::printDevContentOnDev, (void* pDev, Nd4jLong len, int tid), LIBND4J_TYPES);

////////////////////////////////////////////////////////////////////////
template<typename T>
void PointersManager::printDevContentOnHost(const void* pDev, const Nd4jLong len) const {
    printf("host print out\n");
    void* pHost = operator new(sizeof(T) * len);

    hipMemcpyAsync(pHost, pDev, sizeof(T) * len, hipMemcpyDeviceToHost, *_context->getCudaStream());
    hipError_t cudaResult = hipStreamSynchronize(*_context->getCudaStream());
    if(cudaResult != 0)
        throw std::runtime_error("PointersManager::printCudaHost: hipStreamSynchronize failed!");

    for(Nd4jLong i = 0; i < len; ++i)
        printf("%f, ", (double)reinterpret_cast<T*>(pHost)[i]);
    printf("\n");

    operator delete(pHost);
}


template void PointersManager::printDevContentOnHost<Nd4jLong>(const void* pDev, const Nd4jLong len) const;
template void PointersManager::printDevContentOnHost<int>(const void* pDev, const Nd4jLong len) const;
template void PointersManager::printDevContentOnHost<float>(const void* pDev, const Nd4jLong len) const;
template void PointersManager::printDevContentOnHost<double>(const void* pDev, const Nd4jLong len) const;


}
